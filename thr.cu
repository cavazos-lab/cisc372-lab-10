#include "hip/hip_runtime.h"
// Time-stamp: </w/parallel/code/potential/thr.cu, Sun,  5 Apr 2015, 17:25:16 EDT, http://wrfranklin.org/>

// Find the potential energy of N 2D points arranged in a square
// grid.  There are K=N(N-1)/2 pairs of points.  Index the pairs
// from 0 to K-1.  Function k2pot finds the potential energy of
// pair #K.

// Thrust apparently cannot use long long int.

#define PRINTTIMESHORT
#define COLORTTY
#define REALLYPRINTTIME

#include "mycuda.h"

using namespace std;
using namespace thrust;

typedef double Tcoord;

//const int n = 60000;
const int n = 10;

int n1 = sqrt(Tcoord(n))+1;  // # points on a side of the grid

// Make the points in a square grid. 
// Given the index of a point in the grid, find its X and Y coords.
#define coord(n1,k) (thrust::pair<Tcoord,Tcoord> ((k)/(n1), (k)%(n1)))

#define sqr(x) ((x)*(x))

// Given the index of a pair of points, find their potential energy.
struct k2pot : public thrust::unary_function<int,Tcoord>
{
  const int m;
  k2pot(int _m) : m(_m) {}
  __host__ __device__
  Tcoord operator()(const int k) const { 
    int j = sqrt(2.0*k+0.25)+0.5000000001;
    int i = k - j*(j-1LL)/2;
    return (1.0/sqrt(sqr(coord(m,i).first-coord(m,j).first)+sqr(coord(m,i).second-coord(m,j).second)));
  }
};

int main() {
  Delta_Clock_Time();
  cout.imbue(locale("en_US.utf8"));  // Print big numbers with commas.
  cerr.imbue(locale("en_US.utf8"));  // Print big numbers with commas.

  long long int npair = n*(n-1LL)/2;  

#if 0
  cout << "dx dy:" << endl;
  for (int i=0;i<n;i++)
    cout << i << ' ' << coord(n1,i).first << ' ' << coord(n1,i).second << endl;

  cout << "k->i j pot:" << endl;
  for(int k=0; k<npair; k++) {
    int j = sqrt(2.0*k+0.25)+0.5000000001;
    int i = k - j*(j-1LL)/2;
    cout << k << ' ' << i << ' ' << j << ' ' << k2pot(n1)(k) << endl;
  }

#endif

  Tcoord tpot=
    thrust::transform_reduce(counting_iterator<int>(0LL),
			   counting_iterator<int>(npair), k2pot(n1), 0.0,
			   thrust::plus<Tcoord>());

  cout << PRINTC(n) << PRINTC(tpot);
  cout << PRINTN(Delta_Clock_Time());
}

